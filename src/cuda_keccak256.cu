#include "hip/hip_runtime.h"
/**
 * KECCAK-256 CUDA optimised implementation, based on ccminer-alexis code
 */

extern "C" {
#include <fcntl.h>
#include <stdint.h>
#include <memory.h>
#include <stdio.h>
#include <stdlib.h>
}

#include <cuda_helper.h>
#include <cuda_vectors.h>

#define TPB52 1024
#define TPB50 384
#define NPT 2
#define NBN 2

short device_map[MAX_GPUS] = { 0 };
static bool init[MAX_GPUS] = { 0 };

static uint32_t *d_nonces[MAX_GPUS];
static uint32_t *h_nonces[MAX_GPUS];
uint2 highTarget;
FILE* log_fd;

__constant__ uint2 c_message48[6];
__constant__ uint2 c_mid[17];

__constant__ uint2 keccak_round_constants[24] = {
	{ 0x00000001, 0x00000000 }, { 0x00008082, 0x00000000 },	{ 0x0000808a, 0x80000000 }, { 0x80008000, 0x80000000 },
	{ 0x0000808b, 0x00000000 }, { 0x80000001, 0x00000000 },	{ 0x80008081, 0x80000000 }, { 0x00008009, 0x80000000 },
	{ 0x0000008a, 0x00000000 }, { 0x00000088, 0x00000000 },	{ 0x80008009, 0x00000000 }, { 0x8000000a, 0x00000000 },
	{ 0x8000808b, 0x00000000 }, { 0x0000008b, 0x80000000 },	{ 0x00008089, 0x80000000 }, { 0x00008003, 0x80000000 },
	{ 0x00008002, 0x80000000 }, { 0x00000080, 0x80000000 },	{ 0x0000800a, 0x00000000 }, { 0x8000000a, 0x80000000 },
	{ 0x80008081, 0x80000000 }, { 0x00008080, 0x80000000 },	{ 0x80000001, 0x00000000 }, { 0x80008008, 0x80000000 }
};

__device__ __forceinline__
uint2 xor3x(const uint2 a,const uint2 b,const uint2 c) {
	uint2 result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm ("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result.x) : "r"(a.x), "r"(b.x),"r"(c.x)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
	asm ("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result.y) : "r"(a.y), "r"(b.y),"r"(c.y)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
#else
	result = a^b^c;
#endif
	return result;
}

__device__ __forceinline__
uint2 chi(const uint2 a,const uint2 b,const uint2 c) { // keccak chi
	uint2 result;
#if __CUDA_ARCH__ >= 500 && CUDA_VERSION >= 7050
	asm ("lop3.b32 %0, %1, %2, %3, 0xD2;" : "=r"(result.x) : "r"(a.x), "r"(b.x),"r"(c.x)); //0xD2 = 0xF0 ^ ((~0xCC) & 0xAA)
	asm ("lop3.b32 %0, %1, %2, %3, 0xD2;" : "=r"(result.y) : "r"(a.y), "r"(b.y),"r"(c.y)); //0xD2 = 0xF0 ^ ((~0xCC) & 0xAA)
#else
	result = a ^ (~b) & c;
#endif
	return result;
}

__device__ __forceinline__
uint64_t xor5(uint64_t a, uint64_t b, uint64_t c, uint64_t d, uint64_t e)
{
	uint64_t result;
	asm("xor.b64 %0, %1, %2;" : "=l"(result) : "l"(d) ,"l"(e));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(c));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(b));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(a));
	return result;
}

#if __CUDA_ARCH__ <= 500
__global__ __launch_bounds__(TPB50, 2)
#else
__global__ __launch_bounds__(TPB52, 1)
#endif
void keccak256_gpu_hash_80(uint32_t threads, uint32_t startNonce, uint32_t *resNounce, uint2 Target)
{
	uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
	uint2 s[25], t[5], v, w, u[5];
#if __CUDA_ARCH__ > 500
	uint64_t step     = gridDim.x * blockDim.x;
	uint64_t maxNonce = startNonce + threads;
	for(uint64_t nounce = startNonce + thread; nounce<maxNonce;nounce+=step) {
#else
	uint32_t nounce = startNonce+thread;
	if(thread<threads) {
#endif
		s[ 9] = make_uint2(c_message48[0].x,cuda_swab32(nounce));
		s[10] = keccak_round_constants[0];

		t[ 4] = c_message48[1]^s[ 9];
		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		u[ 0] = t[4] ^ c_mid[ 0];
		u[ 1] = c_mid[ 1] ^ ROL2(t[4],1);
		u[ 2] = c_mid[ 2];
		/* thetarho pi: b[..] = rotl(a[..] ^ d[...], ..)*/
		s[ 7] = ROL2(s[10]^u[0], 3);
		s[10] = c_mid[ 3];
		    w = c_mid[ 4];
		s[20] = c_mid[ 5];
		s[ 6] = ROL2(s[ 9]^u[2],20);
		s[ 9] = c_mid[ 6];
		s[22] = c_mid[ 7];
		s[14] = ROL2(u[0],18);
		s[ 2] = c_mid[ 8];
		s[12] = ROL2(u[1],25);
		s[13] = c_mid[ 9];
		s[19] = ROR8(u[1]);
		s[23] = ROR2(u[0],23);
		s[15] = c_mid[10];
		s[ 4] = c_mid[11];
		s[24] = c_mid[12];
		s[21] = ROR2(c_message48[2]^u[1], 9);
		s[ 8] = c_mid[13];
		s[16] = ROR2(c_message48[3]^u[0],28);
		s[ 5] = ROL2(c_message48[4]^u[1],28);
		s[ 3] = ROL2(u[1],21);
		s[18] = c_mid[14];
		s[17] = c_mid[15];
		s[11] = c_mid[16];

		/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
		v = c_message48[5]^u[0];
		s[ 0] = chi(v,w,s[ 2]);
		s[ 1] = chi(w,s[ 2],s[ 3]);
		s[ 2] = chi(s[ 2],s[ 3],s[ 4]);
		s[ 3] = chi(s[ 3],s[ 4],v);
		s[ 4] = chi(s[ 4],v,w);
		v = s[ 5]; w = s[ 6]; s[ 5] = chi(v,w,s[ 7]); s[ 6] = chi(w,s[ 7],s[ 8]); s[ 7] = chi(s[ 7],s[ 8],s[ 9]); s[ 8] = chi(s[ 8],s[ 9],v);s[ 9] = chi(s[ 9],v,w);
		v = s[10]; w = s[11]; s[10] = chi(v,w,s[12]); s[11] = chi(w,s[12],s[13]); s[12] = chi(s[12],s[13],s[14]); s[13] = chi(s[13],s[14],v);s[14] = chi(s[14],v,w);
		v = s[15]; w = s[16]; s[15] = chi(v,w,s[17]); s[16] = chi(w,s[17],s[18]); s[17] = chi(s[17],s[18],s[19]); s[18] = chi(s[18],s[19],v);s[19] = chi(s[19],v,w);
		v = s[20]; w = s[21]; s[20] = chi(v,w,s[22]); s[21] = chi(w,s[22],s[23]); s[22] = chi(s[22],s[23],s[24]); s[23] = chi(s[23],s[24],v);s[24] = chi(s[24],v,w);

		/* iota: a[0,0] ^= round constant */
		s[ 0] ^=keccak_round_constants[ 0];

		#if __CUDA_ARCH__ > 500
			#pragma unroll 22
		#else
			#pragma unroll 4
		#endif
		for (int i = 1; i < 23; i++) {
			#pragma unroll
			for(int j=0;j<5;j++) {
				t[ j] = vectorize(xor5(devectorize(s[ j]),devectorize(s[j+5]),devectorize(s[j+10]),devectorize(s[j+15]),devectorize(s[j+20])));
			}
			/*theta*/
			#pragma unroll
			for(int j=0;j<5;j++) {
				u[j] = ROL2(t[j], 1);
			}
			s[ 4] = xor3x(s[ 4], t[3], u[0]);s[ 9] = xor3x(s[ 9], t[3], u[0]);s[14] = xor3x(s[14], t[3], u[0]);s[19] = xor3x(s[19], t[3], u[0]);s[24] = xor3x(s[24], t[3], u[0]);
			s[ 0] = xor3x(s[ 0], t[4], u[1]);s[ 5] = xor3x(s[ 5], t[4], u[1]);s[10] = xor3x(s[10], t[4], u[1]);s[15] = xor3x(s[15], t[4], u[1]);s[20] = xor3x(s[20], t[4], u[1]);
			s[ 1] = xor3x(s[ 1], t[0], u[2]);s[ 6] = xor3x(s[ 6], t[0], u[2]);s[11] = xor3x(s[11], t[0], u[2]);s[16] = xor3x(s[16], t[0], u[2]);s[21] = xor3x(s[21], t[0], u[2]);
			s[ 2] = xor3x(s[ 2], t[1], u[3]);s[ 7] = xor3x(s[ 7], t[1], u[3]);s[12] = xor3x(s[12], t[1], u[3]);s[17] = xor3x(s[17], t[1], u[3]);s[22] = xor3x(s[22], t[1], u[3]);
			s[ 3] = xor3x(s[ 3], t[2], u[4]);s[ 8] = xor3x(s[ 8], t[2], u[4]);s[13] = xor3x(s[13], t[2], u[4]);s[18] = xor3x(s[18], t[2], u[4]);s[23] = xor3x(s[23], t[2], u[4]);
			/*rho pi: b[..] = rotl(a[..] ^ d[...], ..)*/
			v = s[ 1];
			s[ 1] = ROL2(s[ 6],44);	s[ 6] = ROL2(s[ 9],20);	s[ 9] = ROL2(s[22],61);	s[22] = ROL2(s[14],39);
			s[14] = ROL2(s[20],18);	s[20] = ROL2(s[ 2],62);	s[ 2] = ROL2(s[12],43);	s[12] = ROL2(s[13],25);
			s[13] = ROL8(s[19]);	s[19] = ROR8(s[23]);	s[23] = ROL2(s[15],41);	s[15] = ROL2(s[ 4],27);
			s[ 4] = ROL2(s[24],14);	s[24] = ROL2(s[21], 2);	s[21] = ROL2(s[ 8],55);	s[ 8] = ROL2(s[16],45);
			s[16] = ROL2(s[ 5],36);	s[ 5] = ROL2(s[ 3],28);	s[ 3] = ROL2(s[18],21);	s[18] = ROL2(s[17],15);
			s[17] = ROL2(s[11],10);	s[11] = ROL2(s[ 7], 6);	s[ 7] = ROL2(s[10], 3);	s[10] = ROL2(v, 1);
			/* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
			#pragma unroll
			for(int j=0;j<25;j+=5) {
				v=s[j];w=s[j + 1];s[j] = chi(s[j],s[j+1],s[j+2]);s[j+1] = chi(s[j+1],s[j+2],s[j+3]);s[j+2]=chi(s[j+2],s[j+3],s[j+4]);s[j+3]=chi(s[j+3],s[j+4],v);s[j+4]=chi(s[j+4],v,w);
			}
			/* iota: a[0,0] ^= round constant */
			s[ 0] ^=keccak_round_constants[ i];
		}
		/* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
		#pragma unroll 5
		for(int j=0;j<5;j++) {
			t[ j] = xor3x(xor3x(s[j+0],s[j+5],s[j+10]), s[j+15], s[j+20]);
		}
		s[24] = xor3x(s[24],t[3],ROL2(t[0],1));
		s[18] = xor3x(s[18],t[2],ROL2(t[4],1));
		s[ 0] = xor3x(s[ 0],t[4],ROL2(t[1],1));
		/* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
		s[24] = ROL2(s[24],14);
		s[18] = ROL2(s[18],21);
		if (devectorize(chi(s[18],s[24],s[ 0])) <= devectorize(Target)) {
//		if(chi(s[18].x,s[24].x,s[0].x)<=Target.x) {
//			if(chi(s[18].y,s[24].y,s[0].y)<=Target.y) {
				const uint32_t tmp = atomicExch(&resNounce[0], nounce);
				if (tmp != UINT32_MAX)
					resNounce[1] = tmp;
	//			return;
//			}
		}
	}
}

extern "C" __host__ void keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t* resNonces)
{
	uint32_t tpb;
	dim3 grid;
    /*
	if (device_sm[device_map[thr_id]] <= 500) {
		tpb = TPB50;
		grid.x = (threads + tpb-1)/tpb;
	} else {
    */
		tpb = TPB52;
		grid.x = (threads + (NPT*tpb)-1)/(NPT*tpb);
	//}
	const dim3 block(tpb);

	keccak256_gpu_hash_80<<<grid, block>>>(threads, startNonce, d_nonces[thr_id], highTarget);
//	hipDeviceSynchronize();
	hipMemcpy(h_nonces[thr_id], d_nonces[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);
	memcpy(resNonces, h_nonces[thr_id], NBN*sizeof(uint32_t));
}

__host__
void keccak256_setBlock_80(uint64_t *endiandata)
{
	uint64_t midstate[17], s[25];
	uint64_t t[5], u[5];

	s[10] = 1; //(uint64_t)make_uint2(1, 0);
	s[16] = ((uint64_t)1)<<63; //(uint64_t)make_uint2(0, 0x80000000);

	t[0] = endiandata[0] ^ endiandata[5] ^ s[10];
	t[1] = endiandata[1] ^ endiandata[6] ^ s[16];
	t[2] = endiandata[2] ^ endiandata[7];
	t[3] = endiandata[3] ^ endiandata[8];

	midstate[ 0] = ROTL64(t[1], 1);         //u[0] -partial
	       u[1] = t[ 0] ^ ROTL64(t[2], 1);  //u[1]
	       u[2] = t[ 1] ^ ROTL64(t[3], 1);  //u[2]
	midstate[ 1] = t[ 2];                   //u[3] -partial
	midstate[ 2] = t[ 3] ^ ROTL64(t[0], 1); //u[4]
	midstate[ 3] = ROTL64(endiandata[1]^u[1], 1); //v
	midstate[ 4] = ROTL64(endiandata[6]^u[1], 44);
	midstate[ 5] = ROTL64(endiandata[2]^u[2], 62);
	midstate[ 6] = ROTL64(u[2], 61);
	midstate[ 7] = ROTL64(midstate[2], 39);
	midstate[ 8] = ROTL64(u[2], 43);
	midstate[ 9] = ROTL64(midstate[2], 8);
	midstate[10] = ROTL64(endiandata[4]^midstate[ 2],27);
	midstate[11] = ROTL64(midstate[2], 14);
	midstate[12] = ROTL64(u[1], 2);
	midstate[13] = ROTL64(s[16] ^ u[1], 45);
	midstate[14] = ROTL64(u[2],15);
	midstate[15] = ROTL64(u[1],10);
	midstate[16] = ROTL64(endiandata[7]^u[2], 6);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_mid), midstate,17*sizeof(uint64_t), 0, hipMemcpyHostToDevice));

	// pass only what's needed
	uint64_t message48[6];
	message48[0] = endiandata[9];
	message48[1] = endiandata[4];
	message48[2] = endiandata[8];
	message48[3] = endiandata[5];
	message48[4] = endiandata[3];
	message48[5] = endiandata[0];
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_message48), message48, 6*sizeof(uint64_t), 0, hipMemcpyHostToDevice));
}

__host__
void keccak256_cpu_init(int thr_id)
{
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], NBN*sizeof(uint32_t)));
	//CUDA_SAFE_CALL(hipHostMalloc(&h_nonces[thr_id], NBN*sizeof(uint32_t)));
	h_nonces[thr_id] = (uint32_t*) malloc(NBN * sizeof(uint32_t));
	if(h_nonces[thr_id] == NULL) {
		//gpulog(LOG_ERR,thr_id,"Host memory allocation failed");
		exit(EXIT_FAILURE);
	}
}

__host__
void keccak256_setOutput(int thr_id)
{
	CUDA_SAFE_CALL(hipMemset(d_nonces[thr_id], 0xff, NBN*sizeof(uint32_t)));
}

__host__
void keccak256_cpu_free(int thr_id)
{
	hipFree(d_nonces[thr_id]);
	//hipHostFree(h_nonces[thr_id]);
	free(h_nonces[thr_id]);
}

/* Function to get the compiled Shader Model version */
int cuda_arch[MAX_GPUS] = { 0 };
__global__ void nvcc_get_arch(int *d_version)
{
	*d_version = 0;
#ifdef __CUDA_ARCH__
	*d_version = __CUDA_ARCH__;
#endif
}

__host__
int cuda_get_arch(int thr_id)
{
	int *d_version;
	int dev_id = device_map[thr_id];
	if (cuda_arch[dev_id] == 0) {
		// only do it once...
		hipMalloc(&d_version, sizeof(int));
		nvcc_get_arch <<< 1, 1 >>> (d_version);
		hipMemcpy(&cuda_arch[dev_id], d_version, sizeof(int), hipMemcpyDeviceToHost);
		hipFree(d_version);
	}
	return cuda_arch[dev_id];
}

extern "C" int prepare_mining(uint32_t thr_id, uint32_t throughput, uint64_t* data, uint32_t targetH, uint32_t targetL) {
    log_fd = fopen("gpu.log", "w");
    fprintf(log_fd, "test\n");
	const int dev_id = 0; //device_map[thr_id];
	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
        /*
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
        */
		cuda_get_arch(thr_id);
        keccak256_cpu_init(thr_id);

		init[thr_id] = true;
	}
    fprintf(log_fd, "test2\n");
	highTarget = make_uint2(targetL, targetH);
    keccak256_setBlock_80((uint64_t*)data);
    keccak256_setOutput(thr_id);
    fprintf(log_fd, "test3\n");
    return 0;
}

/*
extern "C" uint32_t mining_iter(uint32_t thr_id, uint32_t throughput, uint32_t first_nonce) {
    *hashes_done = pdata[19] - first_nonce + throughput;
    uint32_t nonces[2] = {UINT32_MAX, UINT32_MAX};
    fprintf(log_fd, "lol");
    keccak256_cpu_hash_80(thr_id, throughput, first_nonce, nonces);
    fprintf(log_fd, "test4\n");
    if (nonces[0] != UINT32_MAX)
    {
        return nonces[0];
    }
    if (nonces[1] != UINT32_MAX)
    {
        return nonces[1];
    }
    return 0;
}
*/

// cleanup
extern "C" void free_keccak256(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

    keccak256_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
