#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h> 
#include <time.h> 
#include <math.h> 


void runBenchmarks();
int gcd(int a, int b);


int clock_speed;
int number_multi_processors;
int number_blocks;
int number_threads;
int max_threads_per_mp;


#define MSG_SIZE 16
#define THREADS_PER_BLOCK 512

const int digest_size = 256;
const int digest_size_bytes = digest_size / 8;

uint64_t h_pre_state[25];    
__device__ uint64_t d_pre_state[25];    
__device__ uint64_t state[25];    

__device__ uint8_t d_message[65];    
__device__ uint64_t d_res_nonces[1];    

// hipEvent_t start, stop;
#define ROTL64(x, y) (((x) << (y)) | ((x) >> (64 - (y))))


__device__ const uint64_t RC[24] = {
    0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
    0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
    0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
    0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
    0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
    0x8000000000008003, 0x8000000000008002, 0x8000000000000080, 
    0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
    0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};

__device__ const int r[24] = {
    1,  3,  6,  10, 15, 21, 28, 36, 45, 55, 2,  14, 
    27, 41, 56, 8,  25, 43, 62, 18, 39, 61, 20, 44
};

__device__ const int piln[24] = {
    10, 7,  11, 17, 18, 3, 5,  16, 8,  21, 24, 4, 
    15, 23, 19, 13, 12, 2, 20, 14, 22, 9,  6,  1 
};

__device__ void keccakF(){
    uint64_t temp, C[5];
	int j;
	
    for (int i = 0; i < 24; i++) {
        // Theta
		// for i = 0 to 5 
		//    C[i] = state[i] ^ state[i + 5] ^ state[i + 10] ^ state[i + 15] ^ state[i + 20];
		C[0] = state[0] ^ state[5] ^ state[10] ^ state[15] ^ state[20];
		C[1] = state[1] ^ state[6] ^ state[11] ^ state[16] ^ state[21];
		C[2] = state[2] ^ state[7] ^ state[12] ^ state[17] ^ state[22];
		C[3] = state[3] ^ state[8] ^ state[13] ^ state[18] ^ state[23];
		C[4] = state[4] ^ state[9] ^ state[14] ^ state[19] ^ state[24];
		
		// for i = 0 to 5
		//     temp = C[(i + 4) % 5] ^ ROTL64(C[(i + 1) % 5], 1);
		//     for j = 0 to 25, j += 5
		//          state[j + i] ^= temp;
		temp = C[4] ^ ROTL64(C[1], 1);
		state[0] ^= temp;
		state[5] ^= temp;
		state[10] ^= temp;
		state[15] ^= temp;
		state[20] ^= temp;
		
		temp = C[0] ^ ROTL64(C[2], 1);
		state[1] ^= temp;
		state[6] ^= temp;
		state[11] ^= temp;
		state[16] ^= temp;
		state[21] ^= temp;
		
		temp = C[1] ^ ROTL64(C[3], 1);
		state[2] ^= temp;
		state[7] ^= temp;
		state[12] ^= temp;
		state[17] ^= temp;
		state[22] ^= temp;
		
		temp = C[2] ^ ROTL64(C[4], 1);
		state[3] ^= temp;
		state[8] ^= temp;
		state[13] ^= temp;
		state[18] ^= temp;
		state[23] ^= temp;
		
		temp = C[3] ^ ROTL64(C[0], 1);
		state[4] ^= temp;
		state[9] ^= temp;
		state[14] ^= temp;
		state[19] ^= temp;
		state[24] ^= temp;
		
        // Rho Pi
		// for i = 0 to 24
		//     j = piln[i];
		//     C[0] = state[j];
		//     state[j] = ROTL64(temp, r[i]);
		//     temp = C[0];
		temp = state[1];
		j = piln[0];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[0]);
		temp = C[0];
		
		j = piln[1];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[1]);
		temp = C[0];
		
		j = piln[2];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[2]);
		temp = C[0];
		
		j = piln[3];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[3]);
		temp = C[0];
		
		j = piln[4];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[4]);
		temp = C[0];
		
		j = piln[5];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[5]);
		temp = C[0];
		
		j = piln[6];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[6]);
		temp = C[0];
		
		j = piln[7];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[7]);
		temp = C[0];
		
		j = piln[8];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[8]);
		temp = C[0];
		
		j = piln[9];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[9]);
		temp = C[0];
		
		j = piln[10];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[10]);
		temp = C[0];
		
		j = piln[11];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[11]);
		temp = C[0];
		
		j = piln[12];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[12]);
		temp = C[0];
		
		j = piln[13];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[13]);
		temp = C[0];
		
		j = piln[14];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[14]);
		temp = C[0];
		
		j = piln[15];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[15]);
		temp = C[0];
		
		j = piln[16];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[16]);
		temp = C[0];
		
		j = piln[17];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[17]);
		temp = C[0];
		
		j = piln[18];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[18]);
		temp = C[0];
		
		j = piln[19];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[19]);
		temp = C[0];
		
		j = piln[20];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[20]);
		temp = C[0];
		
		j = piln[21];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[21]);
		temp = C[0];
		
		j = piln[22];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[22]);
		temp = C[0];
		
		j = piln[23];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[23]);
		temp = C[0];

        //  Chi
		// for j = 0 to 25, j += 5
		//     for i = 0 to 5
		//         C[i] = state[j + i];
		//     for i = 0 to 5
		//         state[j + 1] ^= (~C[(i + 1) % 5]) & C[(i + 2) % 5];
		C[0] = state[0];
		C[1] = state[1];
		C[2] = state[2];
		C[3] = state[3];
		C[4] = state[4];
			
		state[0] ^= (~C[1]) & C[2];
		state[1] ^= (~C[2]) & C[3];
		state[2] ^= (~C[3]) & C[4];
		state[3] ^= (~C[4]) & C[0];
		state[4] ^= (~C[0]) & C[1];
		
		C[0] = state[5];
		C[1] = state[6];
		C[2] = state[7];
		C[3] = state[8];
		C[4] = state[9];
			
		state[5] ^= (~C[1]) & C[2];
		state[6] ^= (~C[2]) & C[3];
		state[7] ^= (~C[3]) & C[4];
		state[8] ^= (~C[4]) & C[0];
		state[9] ^= (~C[0]) & C[1];
		
		C[0] = state[10];
		C[1] = state[11];
		C[2] = state[12];
		C[3] = state[13];
		C[4] = state[14];
			
		state[10] ^= (~C[1]) & C[2];
		state[11] ^= (~C[2]) & C[3];
		state[12] ^= (~C[3]) & C[4];
		state[13] ^= (~C[4]) & C[0];
		state[14] ^= (~C[0]) & C[1];

		C[0] = state[15];
		C[1] = state[16];
		C[2] = state[17];
		C[3] = state[18];
		C[4] = state[19];
			
		state[15] ^= (~C[1]) & C[2];
		state[16] ^= (~C[2]) & C[3];
		state[17] ^= (~C[3]) & C[4];
		state[18] ^= (~C[4]) & C[0];
		state[19] ^= (~C[0]) & C[1];
		
		C[0] = state[20];
		C[1] = state[21];
		C[2] = state[22];
		C[3] = state[23];
		C[4] = state[24];
			
		state[20] ^= (~C[1]) & C[2];
		state[21] ^= (~C[2]) & C[3];
		state[22] ^= (~C[3]) & C[4];
		state[23] ^= (~C[4]) & C[0];
		state[24] ^= (~C[0]) & C[1];
		
        //  Iota
        state[0] ^= RC[i];
    }
}

__global__ void g_set_block() {
    printf("do keccakf\n");
    keccakF();
    printf("done keccakf\n");
}

extern "C" __host__ void h_set_block(const uint8_t *bytes) {
    //get 17 bytes of data, keccakF them
    int rsize = 136;
    int rsize_byte = rsize/8;
    
    memset(h_pre_state, 0, sizeof(h_pre_state));

    for (int i = 0; i < rsize_byte; i++) {
        h_pre_state[i] ^= ((uint64_t *) bytes)[i];
    }
    /*
    printf("pre_state\n");
    for (int i = 0; i < rsize_byte; i++) {
        printf("%d|",h_pre_state[i]);
    }
    printf("\n");
    */
	hipMemcpyToSymbol(HIP_SYMBOL(h_pre_state), d_pre_state, 17*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
	g_set_block<<<1,1>>>();
}

__global__ void g_mine(uint64_t end_nonce, uint64_t target) {
    // get last 64 bytes, pad them and keccakF
    uint8_t temp[144];
    int rsize = 136;
    int rsize_byte = rsize/8;
    int message_len = 64;
    d_res_nonces[0] = UINT64_MAX;
    //printf("do memcpy pre_state\n");
    memcpy(state, d_pre_state, 25);
    // last block and padding
    memcpy(temp, d_message, message_len);
    uint64_t* saltL = ((uint64_t *) temp)+3;
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int num_threads = blockDim.x * gridDim.x;
    //printf("tid %d threads %d\n", tid, num_threads);
    uint64_t start_nonce = *saltL;
    //printf("starting from %d\n", start_nonce+tid);
	while (end_nonce - start_nonce > 0) 
    {
        *saltL = start_nonce+tid;
        temp[message_len] = 0x01;
        memset(temp + message_len, 0, rsize - message_len);
        temp[rsize - 1] |= 0x80;

        for (int i = 0; i < rsize_byte; i++) {
            state[i] ^= ((uint64_t *) temp)[i];
        }
        keccakF();
        if (state[0] <= target) {
            d_res_nonces[0] = start_nonce+tid;
            printf("Cur salt %d|\n", *saltL);
        }
        start_nonce +=  num_threads;
    }
}
extern "C" __host__ uint32_t h_gpu_init(){
    hipDeviceProp_t device_prop;
    int device_count, block_size;

    hipGetDeviceCount(&device_count);
    /*
    if (device_count != 1) {
        exit(EXIT_FAILURE);
    }
    */

    if (hipGetDeviceProperties(&device_prop, 0) != hipSuccess) {
        exit(EXIT_FAILURE);
    } 

    number_threads = device_prop.maxThreadsPerBlock;
    number_multi_processors = device_prop.multiProcessorCount;
    max_threads_per_mp = device_prop.maxThreadsPerMultiProcessor;
    block_size = (max_threads_per_mp / gcd(max_threads_per_mp, number_threads));
    number_threads = max_threads_per_mp / block_size;
    number_blocks = block_size * number_multi_processors;
    clock_speed = (int) (device_prop.memoryClockRate * 1000 * 1000);    
    return number_threads;
}

int gcd(int a, int b) {
    return (a == 0) ? b : gcd(b % a, a);
}

extern "C" __host__ uint64_t h_mine(const uint8_t* message, uint32_t end_nonce, uint64_t target, uint32_t block, uint32_t grid) {
	//dim3 dimBlock(ceil((double)array_size / (double)(512 * 7)));
    dim3 dimBlock(block);
  	dim3 dimGrid(grid);
    uint64_t res_nonces[1] = {UINT64_MAX};

	hipMemcpy(d_message, message, 64*sizeof(uint8_t), hipMemcpyHostToDevice); // copy message to device
	g_mine<<<dimBlock, dimGrid>>>(end_nonce, target);
	hipMemcpy(res_nonces, d_res_nonces, sizeof(uint64_t), hipMemcpyDeviceToHost); // copy message from device
    //hipDeviceSynchronize();
    return res_nonces[0];
}

/*
int main()
{
    h_gpu_init();
    uint8_t test[136];
    memset(test, 1, sizeof(test));
    h_set_block(test);
    dim3 dimBlock(1);
  	dim3 dimGrid(1);
    uint32_t rc = h_mine(test, 0, 10);
    printf("\n%d\n", rc);
    hipDeviceSynchronize();
}
*/
